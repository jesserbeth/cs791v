#include "hip/hip_runtime.h"

#include "kernel.h"

__global__ void reduce(int *in, int *out, int size) {

  // Create Shared Memory
  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int acc = 0;

  // Here is where striding is accomplished
  while(index < size){
    if( index < size){
      acc += in[index];
    }
    // First level reduction: Read from global into shared memory
    if(index + blockDim.x < size){
      acc += in[index+blockDim.x];
    }
    index += blockDim.x * gridDim.x;
  }

    memPnt[tid] = acc;
    __syncthreads();

  for(int j=blockDim.x/2; j > 0; j>>=1){
    if(tid < j){
      memPnt[tid] = acc = acc + memPnt[tid + j];
    }

    __syncthreads();
  }
  if(tid == 0)
    out[blockIdx.x] = acc;

}
