#include "hip/hip_runtime.h"

#include "kernel.h"
#include <stdio.h>

__device__ bool isOdd(int val){
	if(val % 2 == 0){
		return false;
	}
	return true;
}
__device__ int loc_ceil(float val){
  int tmp = (int)val;
  if(tmp == val)
    return val;
  else
    return tmp + 1;
}
__global__ void reduce(int *in, int *out, int size) {

  // Create Shared Memory
  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  // int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int acc = 0;

  // Here is where striding is accomplished
  while(index < size){
    if( index < size){
      acc += in[index];
    }
    // First level reduction: Read from global into shared memory
    // if(index + 1 < size){
    //   acc += in[index+1];
    // }
	   // if(index + blockDim.x < size){
	   //   acc += in[index+blockDim.x];
	   // }
    // if(index + blockDim.x < size){
    //   acc += in[index+blockDim.x];
    // }
    
  //  index += blockDim.x * gridDim.x *2;
    index += blockDim.x * gridDim.x;
 }

    memPnt[tid] = acc;
    __syncthreads();

  // for(int j=blockDim.x/2; j > 0; j>>=1){
  //   if(tid < j){
  //     memPnt[tid] = acc = acc + memPnt[tid + j];
  //   }

  // // for(int j=blockDim.x/2; j > 0; j/=2){
  // //   if(tid < j){
  // //     memPnt[tid] = acc = acc + memPnt[tid + j];
  // //   }
  // //   __syncthreads();
  // }




	// do reduction in shared mem
	// unsigned int r = size/2;
	// bool odd = false;
    // int r = size;
	for(int r = blockDim.x; r > 0; r/=2){
		if(blockDim.x >= r && tid < r/2){
	    	memPnt[tid] = acc = acc + memPnt[tid + r/2];
	    	// memPnt[tid] += memPnt[tid + r/2];
		    if(isOdd(r) && tid == 0)
		    	memPnt[tid] += memPnt[r-1];
		}
		
		__syncthreads();
	}



    // if ((blockDim.x >=   2) && ( tid <  1))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid +  1];
    // }

    // __syncthreads();

	// if (/*(size >= 512) &&*/ (tid < r))
	// // if ((blockDim.x >= 1024) && (tid < 512))
	// {
	//     // memPnt[tid] = acc = acc + memPnt[tid + 512];
	//     memPnt[tid] = acc = acc + memPnt[tid + r*2];
	//     if(isOdd(r) && tid ==0)
	//     	memPnt[tid] += memPnt[r*2-1];
	// }

	// __syncthreads();
	// r /= 2;
	// if (/*(size >= 512) &&*/ (tid < r))
	// // if ((blockDim.x >= 512) && (tid < 256))
	// {
	//     // memPnt[tid] = acc = acc + memPnt[tid + 256];
	//     memPnt[tid] = acc = acc + memPnt[tid + r*2];
	//     if(isOdd(r) && tid ==0)
	//     	memPnt[tid] += memPnt[r*2-1];
	// }

	// __syncthreads();

	// r /= 2;
	// if (/*(size >= 256) &&*/ (tid < r))
	// // if ((blockDim.x >= 256) && (tid < 128))
	// {
	//         // memPnt[tid] = acc = acc + memPnt[tid + 128];
	//         memPnt[tid] = acc = acc + memPnt[tid + r*2];
	//     if(isOdd(r) && tid ==0)
	//     	memPnt[tid] += memPnt[r*2-1];
	// }

	//  __syncthreads();

	// r /= 2;
	// if (/*(size >= 128) &&*/ (tid <  r))
	// // if ((blockDim.x >= 128) && (tid <  64))
	// {
	//    // memPnt[tid] = acc = acc + memPnt[tid +  64];
	//    memPnt[tid] = acc = acc + memPnt[tid +  r*2];
	//     if(isOdd(r) && tid ==0)
	//     	memPnt[tid] += memPnt[r*2-1];
	// }

	// __syncthreads();




    // // fully unroll reduction within a single warp
    // if ((blockDim.x >=  64) && (tid < 32))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid + 32];
    // }

    // __syncthreads();

    // if ((blockDim.x >=  32) && (tid < 16))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid + 16];
    // }

    // __syncthreads();

    // if ((blockDim.x >=  16) && (tid <  8))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid +  8];
    // }

    // __syncthreads();

    // if ((blockDim.x >=   8) && (tid <  4))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid +  4];
    // }

    // __syncthreads();

    // if ((blockDim.x >=   4) && (tid <  2))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid +  2];
    // }

    // __syncthreads();

    // if ((blockDim.x >=   2) && ( tid <  1))
    // {
    //     memPnt[tid] = acc = acc + memPnt[tid +  1];
    // }

    // __syncthreads();

  if(tid == 0){
    // out[blockIdx.x] = acc;
    out[blockIdx.x] = memPnt[tid];
  }

}

__global__ void rReduce(int *in, int *out, int size) {

  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  // int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int acc = 0;

  // Here is where striding is accomplished
  while(index < size){
    if( index < size){
      acc += in[index];
    }
    index += blockDim.x * gridDim.x;
 }

    memPnt[tid] = acc;
    __syncthreads();


  for(int r = blockDim.x; r > 0; r/=2){
    if(blockDim.x >= r && tid < r/2){
        memPnt[tid] = acc = acc + memPnt[tid + r/2];
        if(isOdd(r) && tid == 0)
          memPnt[tid] += memPnt[r-1];
    }
    
    __syncthreads();
  }


  if(tid == 0){
    out[blockIdx.x] = memPnt[tid];
  }

  // Recalculate size
  size = loc_ceil(size / (blockDim.x*2));
  if(size > gridDim.x)
    size = gridDim.x;

  // synchronize across blocks
  __threadfence();

    int memorySize = gridDim.x*2;
    if(gridDim.x <= 32)
      memorySize = 64;

    int *temp = in;
    in = out;
    out = temp;

  // Call next kernel
  if(blockIdx.x*blockDim.x + threadIdx.x == 0 && size >= 1){
      printf("%d, %d\n", tid, size);
      rReduce<<<gridDim.x, blockDim.x, memorySize*sizeof(int)>>>(in, out, size);
  }

  __threadfence();

}