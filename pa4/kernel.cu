#include "hip/hip_runtime.h"

#include "kernel.h"
#include <stdio.h>

__device__ bool isOdd(int val){
	if(val % 2 == 0){
		return false;
	}
	return true;
}
__device__ int loc_ceil(float val){
  int tmp = (int)val;
  if(tmp == val)
    return val;
  else
    return tmp + 1;
}
__global__ void reduce(int *in, int *out, int size) {

  // Create Shared Memory
  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  // int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int acc = 0;

  // Here is where striding is accomplished
  while(index < size){
    if( index < size){
      acc += in[index];
    }
    index += blockDim.x * gridDim.x;
 }

    memPnt[tid] = acc;
    __syncthreads();

	for(int r = blockDim.x; r > 0; r/=2){
		if(blockDim.x >= r && tid < r/2){
	    	memPnt[tid] = acc = acc + memPnt[tid + r/2];
	    	// memPnt[tid] += memPnt[tid + r/2];
		    if(isOdd(r) && tid == 0)
		    	memPnt[tid] += memPnt[r-1];
		}
		
		__syncthreads();
	}


  if(tid == 0){
    // out[blockIdx.x] = acc;
    out[blockIdx.x] = memPnt[tid];
  }

}

__global__ void rReduce(int *in, int *out, int size) {

  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  // int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int acc = 0;

  // Here is where striding is accomplished
  while(index < size){
    if( index < size){
      acc += in[index];
    }
    index += blockDim.x * gridDim.x;
 }

    memPnt[tid] = acc;
    __syncthreads();


  for(int r = blockDim.x; r > 0; r/=2){
    if(blockDim.x >= r && tid < r/2){
        memPnt[tid] = acc = acc + memPnt[tid + r/2];
        if(isOdd(r) && tid == 0)
          memPnt[tid] += memPnt[r-1];
    }
    
    __syncthreads();
  }


  if(tid == 0){
    out[blockIdx.x] = memPnt[tid];
  }

  // Recalculate size
  size = loc_ceil(size / (blockDim.x*2));
  if(size > gridDim.x)
    size = gridDim.x;

  // synchronize across blocks
  __threadfence();

    int memorySize = gridDim.x*2;
    if(gridDim.x <= 32)
      memorySize = 64;

    int *temp = in;
    in = out;
    out = temp;

  // Call next kernel
  if(blockIdx.x*blockDim.x + threadIdx.x == 0 && size >= 1){
      // printf("%d, %d\n", tid, size);
      rReduce<<<gridDim.x, blockDim.x, memorySize*sizeof(int)>>>(in, out, size);
  }

  __threadfence();

}