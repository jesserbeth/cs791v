#include "hip/hip_runtime.h"
/*
  PROGRAMMING ASSIGNMENT 1
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "kernel.h"
#include <fstream> 

int main() {
  // int n, T, B;
  // n = pow(2,21);
  // n = 2097087;
  int check = 0;
  int result = 0;
  float calcTime, memTransTime;
  // char version;
  // std::ofstream out("ParVSSeq.csv");
  // std::ofstream out("test.csv");


// Loop to generate data points (blocks/threads/inputsize)
  for(int n = 1000000; n <= 110000000; n += 20000000){
		char numstr[21];
		sprintf(numstr, "%d", n);
		char f_name[] = "Recursive_Call_";
		char ppm[] = ".csv";
		strcat(f_name,numstr);
		strcat(f_name,ppm);
	  std::ofstream out(f_name);
	  for(int B = 200; B < 56000; B += 4096){
	  	for(int T = 32; T <= 1024; T<<=1){
			  int memorySize = B*2;
			  if(B <= 32)
			  	memorySize = 64;
			  int *input, *output;
			  input = (int*) malloc(n*sizeof(int));
			  output = (int*) malloc(B*sizeof(int));

			  int *g_in, *g_out;

			  hipError_t err = hipMalloc( (void**) &g_in, n * sizeof(int));
			  err = hipMalloc( (void**) &g_out, B * sizeof(int));
			  if (err != hipSuccess) {
			    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			    exit(1);
			  }

			  // Populate Array for addition
			  for (int i = 0; i < n; ++i) {
			    input[i] = 1;
			    // input[i] = i%3;
			  }
			  // Initialize output array to avoid garbage additions
			  for(int i = 0; i < B; i++){
			  	output[i] = 0;
			  }

			  // Create cuda Events
			  hipEvent_t start, end, m_start, m_end;
			  hipEventCreate(&start);
			  hipEventCreate(&end);
			  hipEventCreate(&m_start);
			  hipEventCreate(&m_end);

			  hipEventRecord( m_start, 0 );

			  err = hipMemcpy(g_out, output, B * sizeof(int), hipMemcpyHostToDevice);
			  err = hipMemcpy(g_in, input, n * sizeof(int), hipMemcpyHostToDevice);
			  if (err != hipSuccess) {
			    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			    exit(1);
			  }  

			  // Different calls for each version of code
			  // if(version == 'a'){
				 //  int s = n;
				 //  hipEventRecord(start,0);
				 //  // std::cout << memorySize << std::endl;
				 //  while( s > 1){
				 //    // std::cout << "S IS: " << s << std::endl;
				 //  	// Call Kernel:
				 //  	reduce<<<B,T,B*T*sizeof(int)>>>(g_in, g_out, s);
				  	
				 //  	// Adjust new processing size:
				 //  	s = ceil(s / (T*2));
				 //  	if(s > B)
				 //  		s = B;

				 //  	// Swap pointers to loop
				 //  	int *temp = g_in;
				 //  	g_in = g_out;
				 //  	g_out = temp;

			  //   // 	err = hipMemcpy(output, g_in, B * sizeof(int), hipMemcpyDeviceToHost);
			  //   // 	if (err != hipSuccess) {
			  //   // 	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
			  //   // 	  exit(1);
				 //  	// }
				 //  	// 	std::cout << "OUTPUT VECTOR: " << std::endl;
				 //  	// for(int i = 0; i < B; i++){
				 //  	// 	std::cout << output[i] << std::endl;
				 //  	// }
				 //  }
				 //  hipEventRecord(end, 0);
				 //  hipEventSynchronize(end);

			  //   	err = hipMemcpy(output, g_in, 1 * sizeof(int), hipMemcpyDeviceToHost);
			  //   	if (err != hipSuccess) {
			  //   	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
			  //   	  exit(1);
				 //  	}

				 //  hipEventElapsedTime( &calcTime, start, end );
				  
				 //  hipEventRecord( m_end, 0 );
				 //  hipEventSynchronize( m_end );
				  
				 //  hipEventElapsedTime( &memTransTime, m_start, m_end );

				 //  // Store result to variable
				 //  // result = output[0] + output[1];
				 //  // result = output[0];
				 //  // for(int i = 0; i < B; i++)
				 //  // 	std::cout << "output: " << output[i] << '\n';
			  // hipEventDestroy( start );
			  // hipEventDestroy( end );
			  // hipEventDestroy( m_start );
			  // hipEventDestroy( m_end );

			  // // // write to file
			  // int threads = T;
			  // int blocks = B;
			  // // double memThrough = (n*sizeof(int)) / (memTransTime/1000);
			  // double calcThrough = (n*sizeof(int)*1.0*pow(10,-9)) / (calcTime/1000);
			  // out << calcThrough << ',' << "Gb/s" << ',' << T << ','  << "Threads" << ',' << B << ',' << "Blocks" << ',' << n << '\n' ;
			  // // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
			  // // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
			  // // std::cout << std::endl;
			  // // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
			  // // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
			  
			  // hipFree(g_in);
			  // hipFree(g_out);

			  // free(input);
			  // free(output);
			  // }
			  // else if(version == 'c'){
				  // hipEventRecord(start,0);	
				  // reduce<<<B,T,B*T*sizeof(int)>>>(g_in, g_out, n);
				  // hipEventRecord(end, 0);
				  // hipEventSynchronize(end);

				  // err = hipMemcpy(output, g_out, 1 * sizeof(int), hipMemcpyDeviceToHost);
			   //  	if (err != hipSuccess) {
			   //  	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
			   //  	  exit(1);
				  // 	}
				  // 	// std::cout << "output[0]" << output[0] << std::endl;

				  // result = 0;
				  // for(int i = 0; i < B; i++){
				  // 	result += output[i];
				  // 	// std::cout << output[i] << std::endl;
				  // }
				  // hipEventElapsedTime( &calcTime, start, end );
				  
				  // hipEventRecord( m_end, 0 );
				  // hipEventSynchronize( m_end );
				  
				  // hipEventElapsedTime( &memTransTime, m_start, m_end );

				  // // Correctness check
				  // // result = 0;
				  // // for(int i = 0; i < B; i++){
				  // // 	result += output[i];
				  // // 	// std::cout << output[i] << std::endl;
				  // // }

				  // hipEventDestroy( start );
				  // hipEventDestroy( end );
				  // hipEventDestroy( m_start );
				  // hipEventDestroy( m_end );

				  // // // write to file
				  // int threads = T;
				  // int blocks = B;
				  // // double memThrough = (n*sizeof(int)) / (memTransTime/1000);
				  // double calcThrough = (n*sizeof(int)*1.0*pow(10,-9)) / (calcTime/1000);
				  // out << calcThrough << ',' << "Gb/s" << ',' << T << ','  << "Threads" << ',' << B << ',' << "Blocks" << ',' << n << '\n' ;
				  // // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
				  // // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
				  // // std::cout << std::endl;
				  // // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
				  // // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
				  
				  // hipFree(g_in);
				  // hipFree(g_out);

				  // free(input);
				  // free(output);
			  // }
			  // else if(version == 'b'){
			  	// Recursive Kernel
				  hipEventRecord(start,0);	
				  rReduce<<<B,T,B*T*sizeof(int)>>>(g_in, g_out, n);
				  hipEventRecord(end, 0);
				  hipEventSynchronize(end);


				  err = hipMemcpy(output, g_in, 1 * sizeof(int), hipMemcpyDeviceToHost);
			    	if (err != hipSuccess) {
			    	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
			    	  exit(1);
				  	}
				  	// std::cout << "output[0]" << output[0] << std::endl;

				  result = 0;
				  for(int i = 0; i < B; i++){
				  	result += output[i]; 
				  	// std::cout << output[i] << std::endl;
				  }
				  hipEventElapsedTime( &calcTime, start, end );
				  
				  hipEventRecord( m_end, 0 );
				  hipEventSynchronize( m_end );
				  
				  hipEventElapsedTime( &memTransTime, m_start, m_end );

				  // Correctness check
				  // Store result to variable
				  // result = output[0] + output[1];
				  // result = output[0];
				  // for(int i = 0; i < B; i++)
				  // 	std::cout << "output: " << output[i] << '\n';

				  hipEventDestroy( start );
				  hipEventDestroy( end );
				  hipEventDestroy( m_start );
				  hipEventDestroy( m_end );

				  // // write to file
				  int threads = T;
				  int blocks = B;
				  // double memThrough = (n*sizeof(int)) / (memTransTime/1000);
				  double calcThrough = (n*sizeof(int)*1.0*pow(10,-9)) / (calcTime/1000);
				  out << calcThrough << ',' << "Gb/s" << ',' << T << ','  << "Threads" << ',' << B << ',' << "Blocks" << ',' << n << '\n' ;
				  // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
				  // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
				  // std::cout << std::endl;
				  // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
				  // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
				  
				  hipFree(g_in);
				  hipFree(g_out);

				  free(input);
				  free(output);
			  // }

			  // check = 0;
			  // for(int i = 0; i < n; i++){
			  // 	check += input[i];
			  // }

			  // if(check != result){
			  // 	std::cerr << "Oh no! Something went wrong. You Suck. :(" << std::endl;
			  // 	std::cerr << result << " " << check << std::endl;
			  // 	// std::cerr << result - check << std::endl;
			  // 	hipFree(g_out);
			  // 	hipFree(g_in);
			  // 	free(input);
			  // 	free(output);
			  // 	exit(1);
			  // }
			  
			  // std::cout << "Yay! Your program's results are correct." << std::endl;
			  // std::cout << std::endl;
			  // std::cout << "Your program took: " << memTransTime/1000 << " seconds With Memory Transfer on " << n << " inputs" << std::endl;
			  // std::cout << "Your program took: " << calcTime/1000 << " seconds Without Memory Transfer on " <<  n << " inputs" << std::endl;
			  
			  // Cleanup in the event of success.

		  // out.close();
		  // }


	  	}
	  	// printf("\n");
	  }
	  	printf("%d,\n",n);
	  	out.close();
  }
}	