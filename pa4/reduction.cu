#include "hip/hip_runtime.h"
/*
  PROGRAMMING ASSIGNMENT 1
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "kernel.h"
#include <fstream> 

int main() {
  int n, T, B;
  // n = pow(2,21);
  // n = 2097087;
  int check = 0;
  int result = 0;
  float calcTime, memTransTime;
  char version;
  // std::ofstream out("ParVSSeq.csv");
  // std::ofstream out("test.csv");

  // Handle I/O from user
  printf("Input program version (a, b, c):\na: Multiple cpu kernel calls\nb: Kernel calls kernel\nc: CPU finishes the sum\n");
  scanf(" %c", &version);
  if(version != 'a' && version != 'b' && version != 'c'){
  	printf("Error: Invalid input.\n");
  	return 0;
  }

  // Input size for N, B and T
  printf("Input Size of N (0 < N < 100,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > 100000000){
		printf("Error: Size too large, setting N = 100,000,000 \n");
		n = 100000000;
	}

	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > 1024){
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;
	}

	printf("Input Size of B (0 < B < 1024): ");
	scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > 56000){
		printf("Error: Size too large");
		B = 56000;
	}

	  int memorySize = B*2;
	  if(B <= 32)
	  	memorySize = 64;
	  int *input, *output;
	  input = (int*) malloc(n*sizeof(int));
	  output = (int*) malloc(B*sizeof(int));

	  int *g_in, *g_out;

	  hipError_t err = hipMalloc( (void**) &g_in, n * sizeof(int));
	  err = hipMalloc( (void**) &g_out, B * sizeof(int));
	  if (err != hipSuccess) {
	    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    exit(1);
	  }

	  // Populate Array for addition
	  for (int i = 0; i < n; ++i) {
	    input[i] = 1;
	    // input[i] = i%3;
	  }
	  // Initialize output array to avoid garbage additions
	  for(int i = 0; i < B; i++){
	  	output[i] = 0;
	  }

	  // Create cuda Events
	  hipEvent_t start, end, m_start, m_end;
	  hipEventCreate(&start);
	  hipEventCreate(&end);
	  hipEventCreate(&m_start);
	  hipEventCreate(&m_end);

	  hipEventRecord( m_start, 0 );

	  err = hipMemcpy(g_out, output, B * sizeof(int), hipMemcpyHostToDevice);
	  err = hipMemcpy(g_in, input, n * sizeof(int), hipMemcpyHostToDevice);
	  if (err != hipSuccess) {
	    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    exit(1);
	  }  

	  // Different calls for each version of code
	  if(version == 'a'){
	  	// n = B * T;
		  int s = n;
		  // int s = B;
		  // int count = 0;
		  hipEventRecord(start,0);
		  // std::cout << memorySize << std::endl;
		  while( s > 1){
		    std::cout << "S IS: " << s << std::endl;
		  	// Call Kernel:
		  	reduce<<<B,T,B*T*sizeof(int)>>>(g_in, g_out, s);
		  	
		  	// Adjust new processing size:
		  	s = ceil(s / (T*2));
		  	if(s > B)
		  		s = B;
		  	// s = (s + (T*2-1)) / (T*2);

		  	// Swap pointers to loop
		  	int *temp = g_in;
		  	g_in = g_out;
		  	g_out = temp;
		  	// Count the number of iterations to determine which variable holds the final sum
		  	// std::cout << count << std::endl;
		  	// s = 0;
		  	// count++;

	    	err = hipMemcpy(output, g_in, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}
		  		std::cout << "OUTPUT VECTOR: " << std::endl;
		  	for(int i = 0; i < B; i++){
		  		std::cout << output[i] << std::endl;
		  	}
		  }
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);

	    	err = hipMemcpy(output, g_in, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}

		  hipEventElapsedTime( &calcTime, start, end );
		  
		  hipEventRecord( m_end, 0 );
		  hipEventSynchronize( m_end );
		  
		  hipEventElapsedTime( &memTransTime, m_start, m_end );

		  // Store result to variable
		  result = output[0] + output[1];
		  // result = output[0];
		  for(int i = 0; i < B; i++)
		  	std::cout << "output: " << output[i] << '\n';
	  }
	  else if(version == 'c'){
		  hipEventRecord(start,0);	
		  reduce<<<B,T,memorySize*sizeof(int)>>>(g_in, g_out, n);
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);

		  err = hipMemcpy(output, g_out, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}
		  	// std::cout << "output[0]" << output[0] << std::endl;

		  result = 0;
		  for(int i = 0; i < B; i++){
		  	result += output[i];
		  	// std::cout << output[i] << std::endl;
		  }
		  hipEventElapsedTime( &calcTime, start, end );
		  
		  hipEventRecord( m_end, 0 );
		  hipEventSynchronize( m_end );
		  
		  hipEventElapsedTime( &memTransTime, m_start, m_end );

		  // Correctness check
		  result = 0;
		  for(int i = 0; i < B; i++){
		  	result += output[i];
		  	std::cout << output[i] << std::endl;
		  }
	  }
	  else if(version == 'b'){
	  	// Recursive Kernel
		  hipEventRecord(start,0);	
		  rReduce<<<B,T,memorySize*sizeof(int)>>>(g_in, g_out, n);
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);


		  err = hipMemcpy(output, g_in, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Out Copy Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}
		  	// std::cout << "output[0]" << output[0] << std::endl;

		  result = 0;
		  for(int i = 0; i < B; i++){
		  	result += output[i]; 
		  	// std::cout << output[i] << std::endl;
		  }
		  hipEventElapsedTime( &calcTime, start, end );
		  
		  hipEventRecord( m_end, 0 );
		  hipEventSynchronize( m_end );
		  
		  hipEventElapsedTime( &memTransTime, m_start, m_end );

		  // Correctness check
		  // Store result to variable
		  result = output[0] + output[1];
		  // result = output[0];
		  for(int i = 0; i < B; i++)
		  	std::cout << "output: " << output[i] << '\n';
	  }

		  // result = 0;
		  // for(int i = 0; i < B; i++){
		  // 	result += output[i];
		  // 	// std::cout << output[i] << std::endl;
		  // }
	  check = 0;
	  for(int i = 0; i < n; i++){
	  	check += input[i];
	  }

	  if(check != result){
	  	std::cerr << "Oh no! Something went wrong. You Suck. :(" << std::endl;
	  	std::cerr << result << " " << check << std::endl;
	  	// std::cerr << result - check << std::endl;
	  	hipFree(g_out);
	  	hipFree(g_in);
	  	free(input);
	  	free(output);
	  	exit(1);
	  }
	  
	  // std::cout << "Yay! Your program's results are correct." << std::endl;
	  std::cout << std::endl;
	  std::cout << "Your program took: " << memTransTime/1000 << " seconds With Memory Transfer on " << n << " inputs" << std::endl;
	  std::cout << "Your program took: " << calcTime/1000 << " seconds Without Memory Transfer on " <<  n << " inputs" << std::endl;
	  
	  // Cleanup in the event of success.
	  hipEventDestroy( start );
	  hipEventDestroy( end );
	  hipEventDestroy( m_start );
	  hipEventDestroy( m_end );

	  // // write to file
	  // int threads = i;
	  // int blocks = j;
	  // double memThrough = n / memTransTime;
	  // double calcThrough = n / calcTime;
	  // out << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
	  // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // std::cout << std::endl;
	  // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
	  // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
	  
	  hipFree(g_in);
	  hipFree(g_out);

	  free(input);
	  free(output);

  // out.close();
  // }
}	
