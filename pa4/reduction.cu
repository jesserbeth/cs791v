#include "hip/hip_runtime.h"
/*
  PROGRAMMING ASSIGNMENT 1
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "kernel.h"
#include <fstream> 

int main() {
  int n, T, B;
  // n = pow(2,21);
  // n = 2097087;
  int check = 0;
  int result = 0;
  float calcTime, memTransTime;
  char version;
  // std::ofstream out("ParVSSeq.csv");
  // std::ofstream out("test.csv");

  // Handle I/O from user
  printf("Input program version (a, b, c):\na: Multiple cpu kernel calls\nb: Kernel calls kernel\nc: CPU finishes the sum\n");
  scanf(" %c", &version);
  if(version != 'a' && version != 'b' && version != 'c'){
  	printf("Error: Invalid input.\n");
  	return 0;
  }

  // Input size for N, B and T
  printf("Input Size of N (0 < N < 10,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > 10000000){
		printf("Error: Size too large, setting N = 10,000,000 \n");
		n = 10000000;
	}


	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > 1024){
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;
	}

	printf("Input Size of B (0 < B < 1024): ");
	scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > 56000){
		printf("Error: Size too large");
		B = 56000;
	}

	
  // for(int p = 8; p < 11; p++){
		// // Create sizes
		// // T = pow(2,8);
  // 		T = pow(2,p);
  // 		// n = pow(2,p);
  // 		// T = 1024;
		// B = (n + (T *2 -1))/ T * 2;

		// if(B > 65535)
		// 	B = 65534;
		// if( (B * T) < n)
		// 	std::cout << "fail" << std::endl;
	  int *input, *output;
	  input = (int*) malloc(n*sizeof(int));
	  output = (int*) malloc(B*sizeof(int));

	  int *g_in, *g_out;

	  hipError_t err = hipMalloc( (void**) &g_in, n * sizeof(int));
	  err = hipMalloc( (void**) &g_out, B * sizeof(int));
	  if (err != hipSuccess) {
	    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    exit(1);
	  }

	  // Populate Array for addition
	  for (int i = 0; i < n; ++i) {
	    input[i] = 1;
	  }

	  // Create cuda Events
	  hipEvent_t start, end, m_start, m_end;
	  hipEventCreate(&start);
	  hipEventCreate(&end);
	  hipEventCreate(&m_start);
	  hipEventCreate(&m_end);

	  hipEventRecord( m_start, 0 );

		err = hipMemcpy(g_out, output, B * sizeof(int), hipMemcpyHostToDevice);
	    err = hipMemcpy(g_in, input, n * sizeof(int), hipMemcpyHostToDevice);
	    if (err != hipSuccess) {
	      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	      exit(1);
	    }  

	  if(version == 'a'){
		  int s = n;
		  hipEventRecord(start,0);	
		  while( s > 1){

		  	reduce<<<B,T,T*sizeof(int)>>>(g_in, g_out, s);
		  	s = ceil(s / (T*2));
		  	printf("%d\n", s);
		  	if(s > B)
		  		s = B;

		  	// Swap pointers to loop
		  	int *temp = g_in;
		  	g_in = g_out;
		  	g_out = temp;
		  }
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);

	    	err = hipMemcpy(output, g_out, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}
		  	std::cout << "output[0]" << output[0] << std::endl;

		  hipEventElapsedTime( &calcTime, start, end );
		  
		  hipEventRecord( m_end, 0 );
		  hipEventSynchronize( m_end );
		  
		  hipEventElapsedTime( &memTransTime, m_start, m_end );
	  }
	  else if(version == 'b'){
		  hipEventRecord(start,0);	
		  reduce<<<B,T,T*sizeof(int)>>>(g_in, g_out, n);
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);

		  
	    	err = hipMemcpy(output, g_out, B * sizeof(int), hipMemcpyDeviceToHost);
	    	if (err != hipSuccess) {
	    	  std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    	  exit(1);
		  	}
		  	std::cout << "output[0]" << output[0] << std::endl;

		  result = 0;
		  for(int i = 0; i < B; i++){
		  	result += output[i];
		  	std::cout << output[i] << std::endl;
		  }
		  hipEventElapsedTime( &calcTime, start, end );
		  
		  hipEventRecord( m_end, 0 );
		  hipEventSynchronize( m_end );
		  
		  hipEventElapsedTime( &memTransTime, m_start, m_end );
	  }
	  else{
	  	// Recursive Kernel
	  	int blah = 0;
	  }




	  // Correctness check
	  result = 0;
	  for(int i = 0; i < B; i++){
	  	result += output[i];
	  	std::cout << output[i] << std::endl;
	  }
	  check = 0;
	  for(int i = 0; i < n; i++){
	  	check += input[i];
	  }

	  if(check != result){
	  	std::cerr << "Oh no! Something went wrong. You Suck. :(" << std::endl;
	  	std::cerr << result << " " << check << std::endl;
	  	// std::cerr << result - check << std::endl;
	  	hipFree(g_out);
	  	hipFree(g_in);
	  	free(input);
	  	free(output);
	  	exit(1);
	  }
	  
	  // std::cout << "Yay! Your program's results are correct." << std::endl;
	  std::cout << std::endl;
	  std::cout << "Your program took: " << memTransTime << " ms. With Memory Transfer on " << n << " inputs" << std::endl;
	  std::cout << "Your program took: " << calcTime << " ms. Without Memory Transfer on" <<  n << " inputs" << std::endl;
	  
	  // Cleanup in the event of success.
	  hipEventDestroy( start );
	  hipEventDestroy( end );
	  hipEventDestroy( m_start );
	  hipEventDestroy( m_end );

	  // // write to file
	  // int threads = i;
	  // int blocks = j;
	  double memThrough = n / memTransTime;
	  double calcThrough = n / calcTime;
	  // out << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
	  // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // std::cout << std::endl;
	  // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
	  // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
	  
	  hipFree(g_in);
	  hipFree(g_out);

	  free(input);
	  free(output);

  // out.close();
  // }
}	
