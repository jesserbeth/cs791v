#include "hip/hip_runtime.h"
/*
  PROGRAMMING ASSIGNMENT 1
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "kernel.h"
#include <fstream> 

int main() {
  int n, T, B;
  // n = pow(2,21);
  // n = 2097087;
  int check = 0;
  int result = 0;
  float calcTime, memTransTime;
  // std::ofstream out("ParVSSeq.csv");
  // std::ofstream out("test.csv");

  // Handle I/O from user

  // Input size for N, B and T
  printf("Input Size of N (0 < N < 10,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > 10000000){
		printf("Error: Size too large, setting N = 10,000,000 \n");
		n = 10000000;
	}


	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > 1024){
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;
	}

	printf("Input Size of B (0 < B < 1024): ");
	scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > 56000){
		printf("Error: Size too large");
		B = 56000;
	}

	
  // for(int p = 8; p < 11; p++){
		// // Create sizes
		// // T = pow(2,8);
  // 		T = pow(2,p);
  // 		// n = pow(2,p);
  // 		// T = 1024;
		// B = (n + (T *2 -1))/ T * 2;

		// if(B > 65535)
		// 	B = 65534;
		// if( (B * T) < n)
		// 	std::cout << "fail" << std::endl;
	  int *input, *output;
	  input = (int*) malloc(n*sizeof(int));
	  output = (int*) malloc(B*sizeof(int));

	  int *g_in, *g_out;

	  hipError_t err = hipMalloc( (void**) &g_in, n * sizeof(int));
	  err = hipMalloc( (void**) &g_out, B * sizeof(int));
	  if (err != hipSuccess) {
	    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	    exit(1);
	  }

	  // Populate Array for addition
	  for (int i = 0; i < n; ++i) {
	    input[i] = 1;
	  }

	  // Create cuda Events
	  hipEvent_t start, end, m_start, m_end;
	  hipEventCreate(&start);
	  hipEventCreate(&end);
	  hipEventCreate(&m_start);
	  hipEventCreate(&m_end);

	  hipEventRecord( m_start, 0 );

	  // int s = B;
		err = hipMemcpy(g_out, output, B * sizeof(int), hipMemcpyHostToDevice);
	    err = hipMemcpy(g_in, input, n * sizeof(int), hipMemcpyHostToDevice);
	    if (err != hipSuccess) {
	      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	      exit(1);
	    }  
	  int s = n;
	  // while( s > 1){

	    hipEventRecord(start,0);
	  	reduce<<<B,T,T*sizeof(int)>>>(g_in, g_out, s);
		  hipEventRecord(end, 0);
		  hipEventSynchronize(end);

    	err = hipMemcpy(output, g_out, B * sizeof(int), hipMemcpyDeviceToHost);
    	if (err != hipSuccess) {
    	  std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    	  exit(1);
	  	}

    	// Calc new in/out sizes


	  // }


	  hipEventElapsedTime( &calcTime, start, end );
	  
	  hipEventRecord( m_end, 0 );
	  hipEventSynchronize( m_end );
	  
	  hipEventElapsedTime( &memTransTime, m_start, m_end );

	  result = 0;
	  for(int i = 0; i < B; i++){
	  	result += output[i];
	  }
	  check = 0;
	  for(int i = 0; i < n; i++){
	  	check += input[i];
	  }

	  if(check != result){
	  	std::cerr << "Oh no! Something went wrong. You Suck. :(" << std::endl;
	  	std::cerr << result << " " << check << std::endl;
	  	// std::cerr << result - check << std::endl;
	  	hipFree(g_out);
	  	hipFree(g_in);
	  	free(input);
	  	free(output);
	  	exit(1);
	  }
	  
	  // std::cout << "Yay! Your program's results are correct." << std::endl;
	  std::cout << std::endl;
	  std::cout << "Your program took: " << memTransTime << " ms. With Memory Transfer on " << n << " inputs" << std::endl;
	  std::cout << "Your program took: " << calcTime << " ms. Without Memory Transfer on" <<  n << " inputs" << std::endl;
	  
	  // Cleanup in the event of success.
	  hipEventDestroy( start );
	  hipEventDestroy( end );
	  hipEventDestroy( m_start );
	  hipEventDestroy( m_end );

	  // // write to file
	  // int threads = i;
	  // int blocks = j;
	  double memThrough = n / memTransTime;
	  double calcThrough = n / calcTime;
	  // out << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // out << memThrough << ',' << calcThrough << ',' << n << '\n' ;
	  // std::cout << memThrough << ',' << calcThrough << ',' << T << ',' << B << '\n' ;
	  // std::cout << std::endl;
	  // std::cout << "Throughput: " << memThrough << " ms. With Memory Transfer on " <<  n << " inputs" << std::endl;
	  // std::cout << "Throughput: " << calcThrough << " ms. Without Memory Transfer on " <<  n << " inputs" << std::endl;
	  
	  hipFree(g_in);
	  hipFree(g_out);

	  free(input);
	  free(output);

  // out.close();
  // }
}	
