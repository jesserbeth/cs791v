#include "hip/hip_runtime.h"
/*
  PROGRAMMING ASSIGNMENT 3
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "kernel.h"
#include <fstream> 

int main() {
	int H = 2000;
	int W = 2000;
	int P = H * W;
	int B = 0;
	int iterations = 1024;
	float calcTime, memTime;

	// Mandelbrot Variables
	double MinRe = -2.0;
	double MaxRe = 1.0;
	double MinIm = -1.2;
	double MaxIm = MinIm + (MaxRe-MinRe)*H/W;
	std::ofstream data_out("parallel_data.csv");

	// For loop to generate plot data
	for(int T = 64; T < 1025; T<<=1){
		B = ceil(P / T);

		char numstr[21];
		char threadNum[21];
		sprintf(numstr, "%d", iterations);
		sprintf(threadNum, "_%d", T);
		char f_name[] = "man_par_";
		char ppm[] = ".ppm";
		strcat(f_name,numstr);
		strcat(f_name,threadNum);
		strcat(f_name,ppm);

		std::ofstream im_out(f_name);

		// Declare img vectors
		int *img;
		img = (int*)malloc(P *sizeof(int));
		// Initialize Image
		for(int i = 0; i < P; i++){
			img[i] = 255;
		}

		int *g_out;

		hipError_t err = hipMalloc( (void**) &g_out, P*sizeof(int));
		if (err != hipSuccess) {
		    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		    exit(1);
	    }

	    // Create cuda events
	    hipEvent_t start, end, m_start, m_end;
	    hipEventCreate(&start);
	    hipEventCreate(&end);
	    hipEventCreate(&m_start);
	    hipEventCreate(&m_end);

	    // Start memory event recording
	    hipEventRecord(m_start, 0);

	    err = hipMemcpy(g_out, img, P * sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
		    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		    exit(1);
	    }

	    // Start the kernel
	    hipEventRecord(start, 0);
	    mandel<<<B,T>>>(g_out, iterations, MinRe, MaxRe, MinIm, MaxIm, W, H);
	    hipEventRecord(end, 0);
	    hipEventSynchronize(end);

	    hipEventElapsedTime( &calcTime, start, end);

	    err = hipMemcpy(img, g_out, P * sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
		    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		    exit(1);
	    }

	    hipEventRecord(m_end, 0);
	    hipEventSynchronize(m_end);

	    hipEventElapsedTime(&memTime, m_start, m_end);

		std::cout << std::endl;
		std::cout << "Your program took: " << memTime << " ms. With Memory Transfer with " << T << " threads and " << B << " blocks" << std::endl;
		std::cout << "Your program took: " << calcTime << " ms. Without Memory Transfer with" <<  T  << " threads and " << B << " blocks" << std::endl;

		hipEventDestroy( start );
		hipEventDestroy( end );
		hipEventDestroy( m_start );
		hipEventDestroy( m_end );

		// Write to file
		data_out << memTime << ',' << calcTime << ',' << T << ',' << B << '\n' ;

		// Write to image:
		im_out << "P2" << '\n';
		im_out << "2000 2000" << '\n';
		im_out << "255" << '\n';

		for(int y = 0; y < W; y++){
			for(int x = 0; x < H; x++){
				im_out << img[y*W + x] << " ";
				// std::cout << img[y*W + x] << " ";
				if(img[y*W + x] == 0)
					std::cout << "Zero\n";
			}
			im_out << "\n";
			// std::cout << "\n";
		}
	  
		hipFree(g_out);

		free(img);
		im_out.close();	
	}
	data_out.close();
}	
