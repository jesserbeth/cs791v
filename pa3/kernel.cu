#include "hip/hip_runtime.h"

#include "kernel.h"
#include <math.h>

__global__ void mandel(int *out, int iterations, float minRe, float maxRe, 
                       float minIm, float maxIm, int rowSize, int colSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < rowSize*colSize){
    unsigned y = tid / rowSize;
    unsigned x = tid - rowSize*y;
    float Re_factor = (maxRe - minRe)/(rowSize - 1);
    float Im_factor = (maxIm - minIm)/(colSize - 1);

    float c_im = maxIm - y*Im_factor;
    float c_re = minRe + x*Re_factor;
    float Z_re = c_re, Z_im = c_im;
    bool inside = true;
    for(unsigned n = 0; n<iterations; n++){
      float Z_re2 = Z_re*Z_re, Z_im2 = Z_im*Z_im;
      if(Z_re2 + Z_im2 > 4){
        inside = false;
        break;
      }
      Z_im = 2*Z_re*Z_im + c_im;
      Z_re = Z_re2 - Z_im2 + c_re;
    }
    if(inside == true)
      out[tid] = 0;
  }
}
