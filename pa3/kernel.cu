#include "hip/hip_runtime.h"

#include "kernel.h"

__global__ void mandel(int *out, int iterations, int minRe, int maxRe, 
                       int minIm, int maxIm, int rowSize, int colSize) {
  // int tid = threadIdx.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int y = tid / rowSize;
  int x = tid - rowSize*y;
  float Re_factor = (maxRe - minRe)/(rowSize - 1);
  float Im_factor = (maxIm - minIm)/(colSize - 1);

  // float Z_im = maxIm - y * Im_factor;
  // float Z_re = maxRe - x * Re_factor;
  float c_im = maxIm - y*Im_factor;
  float c_re = minRe + x*Re_factor;
  float Z_re = c_re, Z_im = c_im;
  bool inside = true;
  for(unsigned n=0; n < iterations; n++){
    float Z_re2 = Z_re*Z_re, Z_im2 = Z_im*Z_im;
    if(Z_re2 + Z_im2 > 4){
      inside = false;
      break;
    }
    Z_im = 2*Z_re*Z_im + Z_im;
    Z_re = Z_re2 - Z_im2 + Z_re;
  }

  if(inside == true && tid < (rowSize*colSize))
    out[tid] = 0;
}
