/*
  This program demonstrates the basics of working with cuda. We use
  the GPU to add two arrays. We also introduce cuda's approach to
  error handling and timing using cuda Events.

  This is the main program. You should also look at the header add.h
  for the important declarations, and then look at add.cu to see how
  to define functions that execute on the GPU.
 */

#include <iostream>
#include "stdio.h"
#include "add.h"

int main() {
  int max_threads = 512;
  // int max_blocks = 
  // Arrays on the host (CPU)
  // int a[N], b[N], c[N];
  int n, T, B;
  // Input size for N, B and T
  printf("Input Size of N: ");
  scanf("%d", &n);
  // printf("N: %d \n", size);
  if(n <= 0){
    printf("Error: Wrong input value for N \n");
    return 0;
  }
  printf("Input Size of T: ");
  scanf("%d", &T);
  if(T <= 0){
    printf("Error: Wrong input value for T \n");
    return 0;
  }
  printf("Input Size of B: ");
  scanf("%d", &B);
  if(B <= 0){
    printf("Error: Wrong input value for B \n");
    return 0;
  }
  
  // Dynamically allocate arrays based on keyboard inputs
  int *a, *b, *c;
  a = (int*) malloc(n*sizeof(int));
  b = (int*) malloc(n*sizeof(int));
  c = (int*) malloc(n*sizeof(int));

  // a = (int*) malloc(n);
  // b = (int*) malloc(n);
  // c = (int*) malloc(n);

  /*
    These will point to memory on the GPU - notice the correspondence
    between these pointers and the arrays declared above.
   */
  int *dev_a, *dev_b, *dev_c;

  hipError_t err = hipMalloc( (void**) &dev_a, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_b, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_c, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.   */

  for (int i = 0; i < n; ++i) {
    a[i] = i;
    b[i] = i;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );

  hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);

  add<<<B,T>>>(dev_a, dev_b, dev_c, n);
  
  hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );

  /*
    Let's check that the results are what we expect.
   */
  for (int i = 0; i < n; ++i) {
    if (c[i] != a[i] + b[i]) {
      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

      // clean up events - we should check for error codes here.
      hipEventDestroy( start );
      hipEventDestroy( end );

      // clean up device pointers - just like free in C. We don't have
      // to check error codes for this one.

      hipFree(dev_a);
      hipFree(dev_b);
      hipFree(dev_c);
      free(a);
      free(b);
      free(c);
      exit(1);
    }
  }

  std::cout << "Yay! Your program's results are correct." << std::endl;
  std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
  
  // Cleanup in the event of success.
  hipEventDestroy( start );
  hipEventDestroy( end );



  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  free(a);
  free(b);
  free(c);
}
