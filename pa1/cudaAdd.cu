/*
  This program demonstrates the basics of working with cuda. We use
  the GPU to add two arrays. We also introduce cuda's approach to
  error handling and timing using cuda Events.

  This is the main program. You should also look at the header add.h
  for the important declarations, and then look at add.cu to see how
  to define functions that execute on the GPU.
 */

#include <iostream>
#include "stdio.h"
#include "add.h"

int main() {
  hipDeviceProp_t prop; 	
  hipError_t dev_err = hipGetDeviceProperties(&prop, 0);
  if (dev_err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(dev_err) << std::endl;
    exit(1);
  }
  // printf("threads: %d \n", prop.maxThreadsPerBlock);
  // printf("blocks: %d \n", prop.maxThreadsDim[0]);
  // printf("grids: %d \n", prop.maxGridSize[0]);
  // printf("blocks: %d \n", prop.maxThreadsDim[1]);
  // printf("grids: %d \n", prop.maxGridSize[1]);
  // printf("blocks: %d \n", prop.maxThreadsDim[2]);
  // printf("grids: %d \n", prop.maxGridSize[2]);
  int max_threads = prop.maxThreadsPerBlock;
  int max_blocks = prop.maxThreadsDim[0];
  int max_input = 10000000;
  // Arrays on the host (CPU)
  // int a[N], b[N], c[N];
  int n, T, B;

  // bool flag = false;
  // Input size for N, B and T
  printf("Input Size of N (0 < N < 10,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > max_input)
		printf("Error: Size too large, setting N = 10,000,000 \n");
		n = 10000000;

	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > max_threads)
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;

	  printf("Input Size of B (0 < B < 1024): ");
	  scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > max_blocks)
		printf("Error: Size too large");

  // Dynamically allocate arrays based on keyboard inputs
  int *a, *b, *c;
  a = (int*) malloc(n*sizeof(int));
  b = (int*) malloc(n*sizeof(int));
  c = (int*) malloc(n*sizeof(int));

  // a = (int*) malloc(n);
  // b = (int*) malloc(n);
  // c = (int*) malloc(n);

  /*
    These will point to memory on the GPU - notice the correspondence
    between these pointers and the arrays declared above.
   */
  int *dev_a, *dev_b, *dev_c;

  hipError_t err = hipMalloc( (void**) &dev_a, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_b, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_c, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.   */

  for (int i = 0; i < n; ++i) {
    a[i] = i;
    b[i] = i;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );

  hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);

  add<<<B,T>>>(dev_a, dev_b, dev_c, n);
  
  hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );

  /*
    Let's check that the results are what we expect.
   */
  for (int i = 0; i < n; ++i) {
    if (c[i] != a[i] + b[i]) {
      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

      // clean up events - we should check for error codes here.
      hipEventDestroy( start );
      hipEventDestroy( end );

      // clean up device pointers - just like free in C. We don't have
      // to check error codes for this one.

      hipFree(dev_a);
      hipFree(dev_b);
      hipFree(dev_c);
      free(a);
      free(b);
      free(c);
      exit(1);
    }
  }

  std::cout << "Yay! Your program's results are correct." << std::endl;
  std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
  
  // Cleanup in the event of success.
  hipEventDestroy( start );
  hipEventDestroy( end );



  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  free(a);
  free(b);
  free(c);
}
