/*
  This program demonstrates the basics of working with cuda. We use
  the GPU to add two arrays. We also introduce cuda's approach to
  error handling and timing using cuda Events.

  This is the main program. You should also look at the header add.h
  for the important declarations, and then look at add.cu to see how
  to define functions that execute on the GPU.
 */

#include <iostream>
#include "stdio.h"
#include "add.h"
#include <fstream> 

int main() {
  hipDeviceProp_t prop; 	
  hipError_t dev_err = hipGetDeviceProperties(&prop, 0);
  if (dev_err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(dev_err) << std::endl;
    exit(1);
  }
  // printf("threads: %d \n", prop.maxThreadsPerBlock);
  // printf("blocks: %d \n", prop.maxThreadsDim[0]);
  // printf("grids: %d \n", prop.maxGridSize[0]);
  // printf("blocks: %d \n", prop.maxThreadsDim[1]);
  // printf("grids: %d \n", prop.maxGridSize[1]);
  // printf("blocks: %d \n", prop.maxThreadsDim[2]);
  // printf("grids: %d \n", prop.maxGridSize[2]);
  int max_threads = prop.maxThreadsPerBlock;
  int max_blocks = prop.maxThreadsDim[0];
  int max_input = 10000000;
  int max_seq = max_threads * max_blocks;
  // printf("MAX SEQ: %d", max_seq);
  bool striding = true;
  // Arrays on the host (CPU)
  // int a[N], b[N], c[N];
  int n, T, B;

  printf("Use striding? (No: 0, Yes: 1): ");
  scanf(" %d", &n);
  if(n == 0)
  	striding = false;
  else if(n == 1)
  	striding = true;
  else{
  	printf("Error, Invalid input. Aborting.\n");
  	return 0;
  }
  /*
  // Input size for N, B and T
  printf("Input Size of N (0 < N < 10,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > max_input)
		printf("Error: Size too large, setting N = 10,000,000 \n");
		n = 10000000;

	// Check for striding need
	if(n > max_seq && striding == false){
		printf("Striding is needed for this input size: Using Striding.\n");
		striding = true;
	}

	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > max_threads)
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;

	  printf("Input Size of B (0 < B < 1024): ");
	  scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > max_blocks)
		printf("Error: Size too large");
  */
  
  std::ofstream out("no_striding_max_input.csv");

  n = 1048575;
  // Dynamically allocate arrays based on keyboard inputs
  int *a, *b, *c;
  a = (int*) malloc(n*sizeof(int));
  b = (int*) malloc(n*sizeof(int));
  c = (int*) malloc(n*sizeof(int));

  // a = (int*) malloc(n);
  // b = (int*) malloc(n);
  // c = (int*) malloc(n);

  /*
    These will point to memory on the GPU - notice the correspondence
    between these pointers and the arrays declared above.
   */
  int *dev_a, *dev_b, *dev_c;

  hipError_t err = hipMalloc( (void**) &dev_a, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_b, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_c, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.   */

  for (int i = 0; i < n; ++i) {
    a[i] = i;
    b[i] = i;
  }

  for(int i = 10; i < 1024; i += 100){
  	for(int j = 10; j < 1024; j += 100){
  		  T = i;
  		  B = j;
  		  n = B * T;

		  hipEvent_t start, end;
		  hipEventCreate(&start);
		  hipEventCreate(&end);

		  err = hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
		  err = hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
		  err = hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);
		 
		  if (err != hipSuccess) {
		    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		    exit(1);
		  }


		  hipEventRecord( start, 0 );

		  add<<<B,T>>>(dev_a, dev_b, dev_c, n, striding);
		  hipEventRecord( end, 0 );
		  hipEventSynchronize( end );

		  hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
		  
		  float elapsedTime;
		  hipEventElapsedTime( &elapsedTime, start, end );
		  

		  /*
		    Let's check that the results are what we expect.
		   */
		  for (int i = 0; i < n; ++i) {
		    if (c[i] != a[i] + b[i]) {
		      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

		      // clean up events - we should check for error codes here.
		      hipEventDestroy( start );
		      hipEventDestroy( end );

		      // clean up device pointers - just like free in C. We don't have
		      // to check error codes for this one.

		      hipFree(dev_a);
		      hipFree(dev_b);
		      hipFree(dev_c);
		      free(a);
		      free(b);
		      free(c);
		      exit(1);
		    }
		  }

		  // std::cout << "Yay! Your program's results are correct." << std::endl;
		  // std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
		  
		  // Cleanup in the event of success.
		  hipEventDestroy( start );
		  hipEventDestroy( end );

		  // write to file
		  int threads = i;
		  int blocks = j;
		  out << elapsedTime << ',' << threads << ',' << blocks << '\n' ;

  	}  	// std::cout << i << std::endl;
  	std::cout << 't' << std::endl;
  }
		  hipFree(dev_a);
		  hipFree(dev_b);
		  hipFree(dev_c);

		  free(a);
		  free(b);
		  free(c);

  out.close();
}	
