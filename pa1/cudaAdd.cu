/*
  PROGRAMMING ASSIGNMENT 1
  JESSICA SMITH
  CS791V
 */

#include <iostream>
#include "stdio.h"
#include "add.h"
#include <fstream> 

int main() {
  hipDeviceProp_t prop; 	
  hipError_t dev_err = hipGetDeviceProperties(&prop, 0);
  if (dev_err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(dev_err) << std::endl;
    exit(1);
  }
  int max_threads = prop.maxThreadsPerBlock;
  int max_blocks = prop.maxThreadsDim[0];
  int max_input = 10000000;
  int max_seq;
  bool striding = true;
  int n, T, B;

  printf("Use striding? (No: 0, Yes: 1): ");
  scanf(" %d", &n);
  if(n == 0)
  	striding = false;
  else if(n == 1)
  	striding = true;
  else{
  	printf("Error, Invalid input. Aborting.\n");
  	return 0;
  }

  // Input size for N, B and T
  printf("Input Size of N (0 < N < 10,000,000): ");
	scanf(" %d", &n);
	// printf("N: %d \n", size);
	if(n <= 0)
		printf("Error: Wrong input value for N \n");
	else if( n > max_input)
		printf("Error: Size too large, setting N = 10,000,000 \n");
		n = 10000000;


	printf("Input Size of T (0 < T < 1024): ");
	scanf(" %d", &T);
	  
	if(T <= 0)
		printf("Error: Wrong input value for T \n");
	else if( T > max_threads)
		printf("Error: Size too large, setting T = 1024 \n");
		T = 1024;

	  printf("Input Size of B (0 < B < 1024): ");
	  scanf(" %d", &B);

	if(B <= 0){
	    printf("Error: Wrong input value for B \n");
	  }
	else if( B > max_blocks)
		printf("Error: Size too large");

	max_seq = B * T;
	// Check for striding need
	if(n > max_seq && striding == false){
		printf("Striding is needed for this input size: Using Striding.\n");
		striding = true;
	}
  
  // std::ofstream out("no_striding_max_input.csv");

  // n = 1048575;
  // Dynamically allocate arrays based on keyboard inputs
  int *a, *b, *c;
  a = (int*) malloc(n*sizeof(int));
  b = (int*) malloc(n*sizeof(int));
  c = (int*) malloc(n*sizeof(int));

  int *dev_a, *dev_b, *dev_c;

  hipError_t err = hipMalloc( (void**) &dev_a, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_b, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  err = hipMalloc( (void**) &dev_c, n * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.   */

  for (int i = 0; i < n; ++i) {
    a[i] = i;
    b[i] = i;
  }

  // for(int i = 10; i < 1024; i += 100){
  // 	for(int j = 10; j < 1024; j += 100){
  		  // T = i;
  		  // B = j;
  		  // n = B * T;

		  hipEvent_t start, end;
		  hipEventCreate(&start);
		  hipEventCreate(&end);

		  hipEventRecord( start, 0 );

		  err = hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
		  err = hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
		  err = hipMemcpy(dev_c, c, n * sizeof(int), hipMemcpyHostToDevice);
		 
		  if (err != hipSuccess) {
		    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		    exit(1);
		  }

		  add<<<B,T>>>(dev_a, dev_b, dev_c, n, striding);

		  hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
		  
		  hipEventRecord( end, 0 );
		  hipEventSynchronize( end );
		  
		  float elapsedTime;
		  hipEventElapsedTime( &elapsedTime, start, end );
		  
		  /*
		    Let's check that the results are what we expect.
		   */
		  for (int i = 0; i < n; ++i) {
		    if (c[i] != a[i] + b[i]) {
		      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

		      // clean up events - we should check for error codes here.
		      hipEventDestroy( start );
		      hipEventDestroy( end );

		      // clean up device pointers - just like free in C. We don't have
		      // to check error codes for this one.

		      hipFree(dev_a);
		      hipFree(dev_b);
		      hipFree(dev_c);
		      free(a);
		      free(b);
		      free(c);
		      exit(1);
		    }
		  }

		  std::cout << "Yay! Your program's results are correct." << std::endl;
		  std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
		  
		  // Cleanup in the event of success.
		  hipEventDestroy( start );
		  hipEventDestroy( end );

		  // // write to file
		  // int threads = i;
		  // int blocks = j;
		  // out << elapsedTime << ',' << threads << ',' << blocks << '\n' ;

  // 	} 
  // 	std::cout << 't' << std::endl;
  // }
		  hipFree(dev_a);
		  hipFree(dev_b);
		  hipFree(dev_c);

		  free(a);
		  free(b);
		  free(c);

  // out.close();
}	
