#include "hip/hip_runtime.h"

#include "kernel.h"

/*
  This is the function that each thread will execute on the GPU. The
  fact that it executes on the device is indicated by the __global__
  modifier in front of the return type of the function. After that,
  the signature of the function isn't special - in particular, the
  pointers we pass in should point to memory on the device, but this
  is not indicated by the function's signature.
 */
__global__ void reduce(int *in, int *out, int size) {

  // Create Shared Memory
  extern __shared__ int memPnt[];

  int tid = threadIdx.x;
  int index = blockIdx.x*blockDim.x*2 + threadIdx.x;
  int acc = 0;

  // First level reduction: Read from global into shared memory
  if( index < size){
    acc = in[index];
  }

  if(index + blockDim.x < size){
    acc += in[index+blockDim.x];
  }

  memPnt[tid] = acc;
  __syncthreads();

  // Warp Reduction
  for(int j=blockDim.x/2; j > 0; j>>=1){
    if(tid < j){
      // acc += memPnt[tid + j];
      memPnt[tid] = acc = acc + memPnt[tid + j];
    }

    __syncthreads();
  }
  if(tid == 0)
    out[blockIdx.x] = acc;
}
