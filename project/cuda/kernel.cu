
#include <hip/hip_runtime.h>
// #include "kernel.h"
// #include <math.h>
// #include <stdio.h>

// const int INF = 999999;

/////////////////////////////////////////////////////////////////////////////
//                              Minimal Time
/////////////////////////////////////////////////////////////////////////////
// __global__ void MinTime(float* ignTime, float* rothData, float* times, 
//                    float* L_n, int size, int rowSize, int colSize){
// __global__ void MT(){
   // // Get thread id + stride
   // /* neighbor's address*/     /* N  NE   E  SE   S  SW   W  NW  NNW NNE NEE SEE SSE SSW SWW NWW*/
   // int nCol[16] =        {  0,  1,  1,  1,  0, -1, -1, -1, -1, 1, 2, 2, 1, -1, -2, -2};
   // int nRow[16] =        {  1,  1,  0, -1, -1, -1,  0,  1, 2, 2, 1, -1, -2, -2, -1, 1};
   
   // int cell = blockIdx.x * blockDim.x + threadIdx.x;
   // int ncell, nrow, ncol, row, col;
   // float ignCell, ignCellN, timeNext, timeNow, ROS;
   // // timeNext = INF;
   // // get timenext and timenow from global memory

   // timeNow = times[1]; // timeNow = timeNext
   // timeNext = INF;
   // while(cell < size){
   //    row = cell / rowSize;
   //    col = cell - rowSize*row;
   //    // Load ignition cell to register
   //    ignCell = ignTime[cell];

   //    // Do atomic update of TimeNext Var (atomicMin)
   //    if(timeNext > ignTime[cell] && ignTime[cell] > timeNow){
   //       atomicExch(&times[1], ignCell);
   //       timeNext = ignCell;
   //    }
   //    else if(ignCell == timeNow){

   //       // Find burning cells
   //       for(int n = 0; n < 16; n++){
   //          // Propagate from burning cells      
   //          nrow = row + nRow[n];
   //          ncol = col + nCol[n];
   //          if ( nrow<0 || nrow>= rowSize || ncol<0 || ncol>=  colSize )
   //             continue;
   //          ncell = ncol + nrow*colSize;
   //          ignCellN = ignTime[ncell];
   //          // printf("%f ", ignCellN);

   //          // If neighbor is unburned
   //          if(ignCellN > timeNow){
   //              // compute ignition time
   //              ROS = rothData[3*cell + 0] * (1.0 - rothData[3*cell + 1]) / 
   //                    (1.0 - rothData[3*cell + 1] * cos(rothData[3*cell + 2] * 3.14159/180));
   //              // ROS = 10.0 * (1.0 - 0.321) / (1.0 - 0.321 * cos(30.f));
   //             // ROS = 0.5;
   //              float ignTimeNew = timeNow + L_n[n] / ROS;
   //              // printf("%f, %f, %f, %f \n", timeNow, L_n[n], ROS, ignTimeNew);
   //              if(ignTimeNew < ignCellN){
   //                // printf("%f ", ignTime[ncell]);
   //                  // ignTime[ncell] = ignTimeNew; // This could cause a race cond.
   //                // atomicMin(&ignTime[ncell], ignTimeNew);
   //                if(ignTimeNew < ignTime[ncell])
   //                   ignTime[ncell] = ignTimeNew;
   //                // float tmp = atomicExch(&ignTime[ncell], ignTimeNew);
   //                // if(tmp < ignTimeNew)
   //                //    atomicExch(&ignTime[ncell], ignTimeNew);
   //                  // printf("%f \n",ignTime[ncell]);
   //              }
   //              if(ignTimeNew < timeNext){
   //                  // timeNext = ignTimeNew;
   //                // printf("%f, %f \n", times[1], ignTimeNew);
   //                atomicExch(&times[1], ignTimeNew);
   //                // printf("%f, %f \n \n", times[1], ignTimeNew);
   //              }
   //          }
   //       }
   //    }

   //    // Do striding
   //    cell += blockDim.x * gridDim.x;
   //    // printf("%d \n", cell);
   // }
   // // printf("%f\n", timeNext);
   // if(timeNext == INF){
   //    // printf("BLAH");
//       end = 1;
//       printf("Kernel: %d\n", end);
//    // }
// }

// #endif


// #if IMT

// #endif


// #if BURNDIST

// #endif