#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "fireSim.h"
#include <sys/time.h>
#include "kernel.h"

// const int INF = 999999;

// __device__ int end;

// __global__ void MT(){

//   end = 1;
//   printf("Kernel: %d\n", end);
// }


int main(){
   for(int T = 64; T < 65; T<<=1){
      // Declare simulation variables
      int cell, row, col, nrow, ncol, ncell;
      // char simType[20];
      std::ofstream fout;

      // Initialize simulator
      fireSim sim(T,T);
      struct timeval start, fin;
      // float pi = 3.14159;
      // float ROS = 0;
      // float superSize = sqrt(pow(sim.cellSize, 2) + pow(sim.cellSize*2, 2));
      /* neighbor's address*/     /* N  NE   E  SE   S  SW   W  NW  NNW NNE NEE SEE SSE SSW SWW NWW*/
      // static int nCol[16] =        {  0,  1,  1,  1,  0, -1, -1, -1, -1, 1, 2, 2, 1, -1, -2, -2};
      // static int nRow[16] =        {  1,  1,  0, -1, -1, -1,  0,  1, 2, 2, 1, -1, -2, -2, -1, 1};
      

    sim.init();
    // cout << "initialized" << endl;
    sim.updateSpreadData();

    // Allocate Roth Data for GPU
    // float* gpuRoth = new float[sim.simDimX*sim.simDimY*3];
    float* gpuRoth;
    gpuRoth = (float*)malloc(sim.simDimX*sim.simDimY*3*sizeof(float));
    float* gpuTime;
    gpuTime = (float*)malloc(sim.simDimX*sim.simDimY*sizeof(float));
    float* timeSteppers;
    timeSteppers = (float*)malloc(2*sizeof(float));
    float* loc_L_n;
    loc_L_n = (float*)malloc(16*sizeof(float));
    // cout << "hello" << endl;
    // for(int k = 0, c = 0; k < sim.simDimX*sim.simDimY*3; k+=3, c++){
    for(int k = 0, cell = 0, tcell = 0; k < sim.simDimX; k++){
      for(int c = 0; c < sim.simDimY; c++, cell+=3, tcell++){
        // cout << cell << endl;
        gpuRoth[cell + 0] = sim.rothData[k][c].x;
        gpuRoth[cell + 1] = sim.rothData[k][c].y;
        gpuRoth[cell + 2] = sim.rothData[k][c].z;
        gpuTime[tcell] = sim.ignTime[tcell];
        // cout << gpuTime[tcell] << " " <<sim.ignTime[tcell] << endl;
        // cout << gpuRoth[cell] <<" " << gpuRoth[cell+1] <<" " << gpuRoth[cell+2] << endl;
        // int ROS = gpuRoth[cell + 0] * (1.0 - gpuRoth[cell + 1]) / 
        //               (1.0 - gpuRoth[cell + 1] * cos(gpuRoth[cell + 2] * 3.14159/180));
        // cout << ROS << endl;
      }
    }
    // Allocate Time data for GPU 
    // float* timeSteppers = new float[2];
    // cout << "CPU: " << endl;
    for(int i = 0; i < 16; i++){
      loc_L_n[i] = sim.L_n[i];
    }
    timeSteppers[0] = 0;
    timeSteppers[1] = 0;


    char simType[20];
    sprintf(simType, "../out/GPU_DEBUG");

    // Allocate Cuda Variables
    float *g_ignTime;
    float* g_rothData;
    float* g_times;
    float* g_L_n;

    hipError_t err = hipMalloc( (void**) &g_ignTime, sim.simDimX*sim.simDimY*sizeof(float));
    err = hipMalloc( (void**) &g_rothData, sim.simDimX*sim.simDimY*3*sizeof(float));
    err = hipMalloc( (void**) &g_times, 2*sizeof(float));
    err = hipMalloc( (void**) &g_L_n, 16*sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
      }


    err = hipMemcpy(g_ignTime, gpuTime, sim.simDimX*sim.simDimY*sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(g_rothData, gpuRoth, sim.simDimX*sim.simDimY*3*sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(g_times, timeSteppers, 2*sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(g_L_n, loc_L_n, 16*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    // Kernel Loop
    int counter = 0;
    // cout << "Test" << endl;
    float t1, t2;
    // terminate = 0;
    cout << "Kicking off Kernels" << endl;
    typeof(end) terminate = -1;
    // while(terminate <= 0){
    while(counter < 561){
      counter++;

      // cout << counter << endl;
      MinTime<<<4,4>>>(g_ignTime, g_rothData, g_times, g_L_n, sim.simDimX*sim.simDimY,
                    sim.simDimX, sim.simDimY);
      // MT<<<1,1>>>();
      hipDeviceSynchronize();
      err = hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(end), sizeof(end), 0, 
                                 hipMemcpyDeviceToHost);
      if (err != hipSuccess) {
          std::cerr << "Error copying from GPU: " << hipGetErrorString(err) << std::endl;
          exit(1);
      }
    }
      cout << counter <<endl;

    // Copy back to device
    err = hipMemcpy(gpuTime, g_ignTime, sim.simDimX*sim.simDimY*sizeof(float), hipMemcpyDeviceToHost);
    // err = hipMemcpy(gpuRoth, g_rothData, sim.simDimX*sim.simDimY*3*sizeof(float), hipMemcpyDeviceToHost);
    // err = hipMemcpy(timeSteppers, g_times, 2*sizeof(float), hipMemcpyDeviceToHost);
    // err = hipMemcpy(sim.L_n, g_L_n, 16*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying from GPU: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

      // Write data to file
      char threadNum[21];
      sprintf(threadNum, "_%d_%d", sim.simDimX, sim.simDimY);
      char csv[] = ".csv";
      strcat(simType,threadNum);
      strcat(simType,csv);
      fout.open(simType);
      for(int i = 0; i < sim.simDimX*sim.simDimY; i++){
        // std::cout << ignTime[i] << " ,";
        if(i %sim.simDimX == 0 && i !=0){
            // std::cout << std::endl;
            fout << '\n';
        }
        // fout << (int)sim.ignTime[i] << " ";
        // fout << (int)ignTimeNew[i] << " ";
        fout << (int) gpuTime[i] << " ";
      }
      fout.close();
      
   }

   return 0;
}